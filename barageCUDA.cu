﻿
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cmath>


#define NthMax 128 //Définit le nombre de threads max/block
#define MAX(a,b) (((a)>(b))?(a):(b))

//fonction fournie
__device__ __forceinline__ double atomMax(double* address, double val)
{
    unsigned long long ret = __double_as_longlong(*address);
    while (val > __longlong_as_double(ret))
    {
        unsigned long long old = ret;
        if ((ret = atomicCAS((unsigned long long*)address, old, __double_as_longlong(val))) == old)
            break;
    }
    return __longlong_as_double(ret);
}


// Operation de reduction depuis le device
// in : tab double *a
// out : *maxi = max(a)
// algo : arbre descendant entre threads (O(log N)) + atomicMax (double)
// mem : stockage en shared des portions de a
__device__ void redMaxDblOptDev(int N, double* a, double* maxi)
{
    __shared__ double tmp[NthMax];
    int idx = blockIdx.x * blockDim.x + threadIdx.x, s;

    if (idx < N)
    {
        tmp[threadIdx.x] = a[idx];
    }
    else
    {
        tmp[threadIdx.x] = 0.;
    }

    __syncthreads();

    for (s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            tmp[threadIdx.x] = MAX(tmp[threadIdx.x], tmp[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        atomMax(maxi, tmp[0]);
    }
}

//A utiliser pour calculer le max du tableau a dans la valeur maxi à partir de code CPU
__global__ void redMaxDblOptKer(int N, double* a, double* maxi)
{
    redMaxDblOptDev(N, a, maxi);
}

// initialisation des champs
void init(int N, double dx, double* x, double* h, double* hu)
{
    int i;
    for (i = 0; i < N; i++)
    {
        x[i] = ((double)i + 0.5) * dx;      // espace
        if (x[i] < 0.5)  h[i] = 10.0;  // hauteur : discontinue en x=0.5
        else          h[i] = 1.0;   //
        hu[i] = 0.0;                // vitesse nulle partout => hu = 0 !

    }
}

/* 
integration connaissant les flux
 f_h et f_hu
Vol. finis : u[i]^(n+1) = u[i]^n - dt( F[i+1/2] - F[i-1/2] )

Paramètres : 
    - N : nombre de points
    - dx : pas de discrétisation
    - cm : célérité max calculée avec la fonction Flux_kernel()
    - h : vecteur taille N de la hauteur à l'instant n
    - hu : vecteur taille N de la hauteur*vitesse à l'instant n
    - fh : vecteur taille N flux hauteur
    - fu : vecteur taille N flux vitesse
*/
__global__ void integre_kernel(int N, double* dx, double* cm, double* h, double* hu, double* fh, double* fu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //On récupère le thread qui va faire le calcul (seulement dans des blocs de dimension 1
    double dt = 0.75 * (*dx) / (*cm); //On calcule le nouveau pas de temps
    double rap = dt / (*dx); //Constante de calcul

    if (i >= 1 && i <= N - 2) { //si on n'est pas au bord calcul normal
        h[i] = h[i] - rap * (fh[i] - fh[i - 1]);
        hu[i] = hu[i] - rap * (fu[i] - fu[i - 1]);
    }
    else if (i == 0) { //Condition limite en 0
        h[0] = h[1];  hu[0] = hu[1]; // gauche => sortie libre : hg = hd, ug = ud
    }
    else { //Condition limite en N-1
        h[N - 1] = h[N - 2]; hu[N - 1] = -hu[N - 2]; // droite => mur : hg = hd, ug = -ud
    }    
}

//Calcule le flux à partir des hg,hd,ug,ud et calcule la valeur max
/*
Paramètres:
    - hg,hd : hauteur gauche/droite
    - ug,ud : vitesse gauche/droite
    - fh : vecteur taille N flux hauteur
    - fu : vecteur taille N flux vitesse
    - cmax : célérité max
*/
__device__ void calculFLux_kernel(double hg, double hd, double ug, double ud, double* fh, double *fu, double *cmax) {
    double g = 9.81, cm, cg, cd, c1, c2;
    cg = sqrt(g * hg);  cd = sqrt(g * hd);

    // Calcul des vitesse d'onde
    c1 = fmin(ug - cg, ud - cd);
    c2 = fmax(ug + cg, ud + cd);

    if (c1 >= 0.0) { // toutes les ondes traversent par la droite
        *fh = hg * ug;
        *fu = hg * ug * ug + 0.5 * g * hg * hg;
        cm = fabs(c2);
    }
    else if (c2 <= 0.0) { // toutes les ondes traversent a gauche
        *fh = hd * ud;
        *fu = hd * ud * ud + 0.5 * g * hd * hd;
        cm = fabs(c1);
    }
    else {   // cas ou l'on a un probleme de Riemann
        // Flux HLL F = (c2*fg - c1*fd)/(c2-c1)  + c1*c2*(Ud - Ug)/(c2-c1)
        *fh = (c2 * hg * ug - c1 * hd * ud) / (c2 - c1) + c1 * c2 * (hd - hg) / (c2 - c1);
        *fu = (c2 * (hg * ug * ug + 0.5 * g * hg * hg) - c1 * (hd * ud * ud + 0.5 * g * hd * hd)) / (c2 - c1) + c1 * c2 * (hd * ud - hg * ug) / (c2 - c1);
        cm = fmax(fabs(c1), fabs(c2));
    }

    *cmax = fmax(*cmax, cm); // vitesse d'onde max
}

/*Résoudre les flux pour chaque point
* Tableau cm_tot prend toutes le valeurs calculées par chaque thread et on utilise la réduction pour trouver la valeur max stockée dans *cmax
Paramètres:
    - N : nombre de points
    - h : vecteur taille N de la hauteur à l'instant n
    - hu : vecteur taille N de la hauteur*vitesse à l'instant n
    - fh : vecteur taille N flux hauteur
    - fu : vecteur taille N flux vitesse
    - cm_tot_d : vecteur taille N qui stocke la célérité max calculée par chaque thread i
    - cmax : célérité max sur laquelle on va faire la réduction du vecteur cm_tot_d
*/
__global__ void Flux_kernel(int N, double* h, double* hu, double* fh, double* fu, double* cm_tot_d, double* cmax)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //On récupère le thread qui va faire le calcul
    
    double g = 9.81, cm, cg, cd, c1, c2;
    double hg, hd, ug, ud;

    if (i <= N - 2) { //Si on est l'avant dernier thread: on fait le calcul de flux

        hg = h[i]; hd = h[i + 1];
        ug = hu[i] / hg; ud = hu[i + 1] / hd;
        calculFLux_kernel(hg, hd, ug, ud, &fh[i], &fu[i], &cm_tot_d[i]); //Calcul du flux et stockage de la vitesse max dans le tableau des vitesses max
    }

    redMaxDblOptDev(N, cm_tot_d, cmax); //Réduction dans cmax
}

// ecriture des résultats dans un fichier
// sauvegarde de h et u
void ecrit(int N, FILE* fichier, double* x, double* h, double* hu)
{
    int i;
    for (i = 0; i < N; i++)
    {
        fprintf(fichier, "%lf %lf %lf\n", x[i], h[i], hu[i] / h[i]);
    }
    fclose(fichier); // fermeture
}

//fonction principale
//Nb de threads par block maxi : 1024
int main(int argc, char* argv[])
{
    int N, Nt; //Nombre de points discrétisation, itérations et nombre maxi de threads/bloc
    int it; //Pour l'itération en temps
    double dx; //Variables CPU
    double *dx_d, *cm_d; //Variables GPU, 

    //Tableaux CPU
    double* x, * h, * hu;

    //Tableaux GPU
    double* x_d, * h_d, * hu_d, * fh_d, * fu_d, * cm_tot_d;

    FILE* fparam, *finit, * fres; //Fichiers de sauvegarde

    fparam = fopen("param.dat", "r");
    fscanf(fparam, "%d %d\n", &N, &Nt);
    fclose(fparam);

    //Dimensions grille/block
    dim3 grid(N/NthMax + 1, 1, 1); 
    dim3 block(NthMax, 1, 1);

    printf("N : %d , Nt : %d\n", N, Nt); //Check paramètres de calcul 
    printf("Nblocks : %d , Nthreads/block : %d\n", N / NthMax + 1, NthMax); //Check répartition threads/blocks

    // ouverture des fichiers
    finit = fopen("initial.dat", "w"); 
    fres = fopen("finale.dat", "w");

    dx = 1.0 / (double)(N); //Pas de discrétisation spatiale

    //Allocation CPU, pas de fu ou fh car on va faire tous les calculs dans le GPU
    x =  (double*)malloc(N * sizeof(double));  //Abcisse
    h =  (double*)malloc(N * sizeof(double)); //Hauteur
    hu = (double*)malloc(N * sizeof(double)); //Hauteur*vitesse


    //Allocation GPU : suffixe "_d" pour "device"
    hipMalloc((void**) &h_d, N * sizeof(double)); 
    hipMalloc((void**)&hu_d, N * sizeof(double)); 
    hipMalloc((void**)&fh_d, N * sizeof(double));
    hipMalloc((void**)&fu_d, N * sizeof(double)); 
    hipMalloc((void**)&cm_tot_d, N * sizeof(double)); //va stocker toutes les valeurs des célérités max de chaque thread
    hipMalloc((void**)&cm_d, sizeof(double)); //Valeur max sur tout le domaine sur laquelle on va effectuer la réduction de cm_tot_d
    hipMalloc((void**)&dx_d, sizeof(double)); //pas d'espace

    init(N, dx, x, h, hu); //Initialisation des champs h et hu 
    ecrit(N, finit, x, h, hu); //Sauvegarde dans le fichier "initial.dat"

    //Copie des variables que l'on a initialisées sur le host sur les variables device
    hipMemcpy(h_d, h, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(hu_d, hu, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dx_d, &dx,sizeof(double), hipMemcpyHostToDevice);

    for (it = 1; it <= Nt; it++)   // boucle en temps
    {
        Flux_kernel <<<grid, block>>> (N, h_d, hu_d, fh_d, fu_d, cm_tot_d, cm_d);     // calcul des flux HLL et vitesse d'onde (cm)
        integre_kernel <<<grid, block>>> (N, dx_d, cm_d, h_d, hu_d, fh_d, fu_d); // integration sur un pas de temps Vol. Finis
    }

    //Récupérer les valeurs de h et hu calculées
    hipMemcpy(h, h_d, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hu, hu_d, N * sizeof(double), hipMemcpyDeviceToHost);

    ecrit(N, fres, x, h, hu); //Ecrire résultats

    free(x); free(h); free(hu);
    hipFree(h_d); hipFree(hu_d); hipFree(fh_d); hipFree(fu_d); hipFree(cm_tot_d); hipFree(cm_d); hipFree(dx_d);
    return 0;
}

